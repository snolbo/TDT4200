#include <stdio.h>
#include <stdlib.h>
#include <sys/time.h>
#include <hip/hip_runtime.h>
#include <math.h>


#define XSIZE 2560
#define YSIZE 2048

#define MAXITER 255
#define PIXEL(i,j) ((i)+(j)*XSIZE)


// Same as PS1
typedef unsigned char uchar;
typedef struct {
	float real;
  float imag;
} complex_t;

// implement these
void calculate_cuda(float x_start, float ylower, float step);
__global__
void julia_kernel(int* pixel_device, float x_start, float ylower, float step);

// utilities
void output_bmp();
double walltime();
void calculate_serial();



float x_start=-2.01;
float x_end=1;
float yupper;
float ylower;
float ycenter=1e-6;
float step;

complex_t julia_num;


int pixel_host[XSIZE*YSIZE];
int pixel[XSIZE*YSIZE];

double walltime() {
    static struct timeval t;
    gettimeofday(&t, NULL);
    return (t.tv_sec + 1e-6 * t.tv_usec);
}


// Set up the cuda memory transfers, launch your kernel and extract the finished image
void calculate_cuda(float x_start, float ylower, float step){
	int* pixel_device;
	hipMalloc(&pixel_device, XSIZE*YSIZE*sizeof(int));
	size_t threads_per_block_dim = 32;
	// Assumin that XSIZE and YSIZE is dividable by 32
	dim3 gridBlock(XSIZE/threads_per_block_dim, YSIZE/threads_per_block_dim);
	dim3 threadBlock(threads_per_block_dim, threads_per_block_dim);
	julia_kernel<<<gridBlock, threadBlock>>>(pixel_device, x_start, ylower, step);
	hipMemcpy(pixel_host, pixel_device, XSIZE*YSIZE*sizeof(int), hipMemcpyDeviceToHost);
  hipFree(pixel_device);

}


// Implement the kernel responsible for iterating a single pixel
__global__
void julia_kernel(int* pixel_device, float x_start, float ylower, float step){
	int i = threadIdx.x + blockIdx.x * blockDim.x;
	int j = threadIdx.y + blockIdx.y * blockDim.y;
	complex_t c, z, temp;
	int iter = 0;
	c.real = (x_start + step * i);
	c.imag = (ylower + step * j);
	z = c;
	while(z.real*z.real + z.imag*z.imag < 4){
		temp.real = z.real*z.real - z.imag*z.imag + c.real;
		temp.imag = 2.0*z.real*z.imag + c.imag;
		z = temp;
		if(++iter==MAXITER) break;
	}
	pixel_device[PIXEL(i,j)] = iter;
}

int main(int argc, char **argv) {

	if(argc==1) {
		puts("Usage: JULIA\n");
		puts("Input real and imaginary part. ex: ./julia 0.0 -0.8");
		return 0;
	}

  julia_num.real = strtod(argv[1], NULL);
  julia_num.imag = strtod(argv[2], NULL);

  /* Calculate the range in the y - axis such that we preserve the aspect ratio */
  step = (x_end - x_start)/XSIZE;
  yupper = ycenter + (step * YSIZE)/2;
  ylower = ycenter - (step * YSIZE)/2;



  printf("Calculating with the serial implementation...\n");
  double start_serial = walltime();
  calculate_serial();
  double end_serial = walltime();
  printf("Computation complete. It took %7.3f ms\n\n\n", end_serial - start_serial);


  printf("Checking GPU(s)\n");

  int n_devices;
  hipGetDeviceCount(&n_devices);
  printf("Number of CUDA devices: %d\n", n_devices);
  hipDeviceProp_t device_prop;
  hipGetDeviceProperties(&device_prop, 0);
  printf("CUDA device name 1: %s\n" , device_prop.name);

  if((n_devices < 1) || (n_devices > 2)){
    printf("You're either on more than 2 GPUs, or something is broken\n");
    printf("Exiting");
    exit(0);
  }

  printf("Calculating with CUDA...\n");
  double start_gpu = walltime();
  calculate_cuda(x_start, ylower, step);
  double end_gpu = walltime();
  printf("Computation complete. It took %7.3f ms\n", end_gpu - start_gpu);


  output_bmp();

  return 0;
}



//////////////////////////////////////////
//////////////////////////////////////////
//////////////////////////////////////////
////// UTILITIES, ALREADY IMPLEMENTED
complex_t add_complex(complex_t a, complex_t b){
  complex_t temp;
  temp.real = a.real + b.real;
  temp.imag = a.imag + b.imag;
  return temp;
}

complex_t add_real(complex_t a, int b){
  complex_t temp;
  temp.real = a.real + b;
  return temp;
}

complex_t square_complex(complex_t c){
  complex_t temp;
  temp.real = c.real*c.real - (c.imag*c.imag);
  temp.imag = 2*c.imag*c.real;
  return temp;
}


void savebmp(char *name,uchar *buffer,int x,int y);
void fancycolour(uchar *p,int iter);

void output_bmp(){
  unsigned char* img_buffer = (unsigned char*)calloc(XSIZE*YSIZE*3, 1);
  for(int ii = 0; ii < XSIZE; ii++){
    for(int jj = 0; jj < YSIZE; jj++){
      int p=((YSIZE-jj-1)*XSIZE+ii)*3;
      fancycolour(img_buffer+p,pixel_host[PIXEL(ii,jj)]);
    }
  }

  char filename[20] = "julia.bmp";
  savebmp(filename, img_buffer, XSIZE, YSIZE);
  free(img_buffer);
}



/* save 24-bits bmp file, buffer must be in bmp format: upside-down */
void savebmp(char *name,uchar *buffer,int x,int y) {
	FILE *f=fopen(name,"wb");
	if(!f) {
		printf("Error writing image to disk.\n");
		return;
	}
	unsigned int size=x*y*3+54;
	uchar header[54]={'B','M',size&255,(size>>8)&255,(size>>16)&255,size>>24,0,
                    0,0,0,54,0,0,0,40,0,0,0,x&255,x>>8,0,0,y&255,y>>8,0,0,1,0,24,0,0,0,0,0,0,
                    0,0,0,0,0,0,0,0,0,0,0,0,0,0,0,0,0,0,0};
	fwrite(header,1,54,f);
	fwrite(buffer,1,XSIZE*YSIZE*3,f);
	fclose(f);
}


/* given iteration number, set a colour */
void fancycolour(uchar *p,int iter) {
	if(iter==MAXITER);
	else if(iter<8) { p[0]=128+iter*16; p[1]=p[2]=0; }
	else if(iter<24) { p[0]=255; p[1]=p[2]=(iter-8)*16; }
	else if(iter<160) { p[0]=p[1]=255-(iter-24)*2; p[2]=255; }
	else { p[0]=p[1]=(iter-160)*2; p[2]=255-(iter-160)*2; }
}


void calculate_serial() {
	for(int i=0;i<XSIZE;i++) {
		for(int j=0;j<YSIZE;j++) {

			/* Calculate the number of iterations until divergence for each pixel.
			   If divergence never happens, return MAXITER */
			complex_t c;
      complex_t z;
      complex_t temp;
			int iter=0;

      // find our starting complex number c
			c.real = (x_start + step*i);
			c.imag = (ylower + step*j);

      // our starting z is c
			z = c;

      // iterate until we escape
			while(z.real*z.real + z.imag*z.imag < 4) {
        temp.real = (z.real*z.real) - (z.imag*z.imag);
        temp.imag = 2*z.real*z.imag;

        temp.real += julia_num.real;
        temp.imag += julia_num.imag;

				z = temp;
				if(++iter==MAXITER) break;
			}
			pixel[PIXEL(i,j)]=iter;
		}
	}
}
