#include "hip/hip_runtime.h"
#include <stdio.h>
#include <stdlib.h>
#include <math.h>
#include <sys/time.h>
#include <hip/hip_runtime.h>
#include <hip/hip_runtime.h>

#define dT 0.2f
#define G 0.6f
#define BLOCK_SIZE 512

// Global variables
int num_planets;
int num_timesteps;

// Host arrays
float2* velocities;
float4* planets;

// Device arrays
float2* velocities_d;
float4* planets_d;


double walltime() {
    static struct timeval t;
    gettimeofday(&t, NULL);
    return (t.tv_sec + 1e-6 * t.tv_usec);
}


// Parse command line arguments
void parse_args(int argc, char** argv){
    if(argc != 2){
        printf("Useage: nbody num_timesteps\n");
        exit(-1);
    }

    num_timesteps = strtol(argv[1], 0, 10);
}

// Reads planets from planets.txt
void read_planets(){

  //char* a;
  FILE* file = fopen("planets256.txt", "r");
  if(file == NULL){
      printf("'planets.txt' not found. Exiting\n");
      exit(-1);
  }

  char line[200];
  fgets(line, 200, file);
  sscanf(line, "%d", &num_planets);

    planets = (float4*)malloc(sizeof(float4)*num_planets);
    velocities = (float2*)malloc(sizeof(float2)*num_planets);

    for(int p = 0; p < num_planets; p++){
        fgets(line, 200, file);
        sscanf(line, "%f %f %f %f %f",
                &planets[p].x,
                &planets[p].y,
                &velocities[p].x,
                &velocities[p].y,
                &planets[p].z);
    }

    fclose(file);
}

// Writes planets to file
void write_planets(int timestep){
    char name[20];
    int n = sprintf(name, "gpu_planets_out.txt");

    FILE* file = fopen(name, "wr+");

    for(int p = 0; p < num_planets; p++){
        fprintf(file, "%f %f %f %f %f\n",
                planets[p].x,
                planets[p].y,
                velocities[p].x,
                velocities[p].y,
                planets[p].z);
    }

    fclose(file);
}

// TODO 7. Calculate the change in velocity for p, caused by the interaction with q
__device__ float2 calculate_velocity_change_planet(float4 p, float4 q){

  float2 dist;
  float2 dvel;
  dist.x = q.x - p.x;
  dist.y = q.y - p.y;
  if(dist.x == 0 && dist.y == 0){
  float2 vel = {0.0f, 0.0f};
  return vel;
  }
  float abs_dist= sqrt(dist.x*dist.x + dist.y*dist.y);
  float dist_cubed = abs_dist*abs_dist*abs_dist;
  // Calculate change in velocity
  dvel.x = dT*G*q.z/dist_cubed * dist.x;
  dvel.y = dT*G*q.z/dist_cubed * dist.y;
  return dvel;
}

// TODO 5. Calculate the change in velocity for my_planet, caused by the interactions with a block of planets
__device__ float2 calculate_velocity_change_block(float4 my_planet, float4* shared_planets){
  float2 velocity;
  velocity.x = 0.0f;
  velocity.y = 0.0f;
  for(int i = 0; i < blockDim.x; i++){
    float2 temp_vel = calculate_velocity_change_planet(my_planet, shared_planets[i]);
    velocity.x += temp_vel.x;
    velocity.y += temp_vel.y;
  }
  return velocity;
}

// TODO 4. Update the velocities by calculating the planet interactions
__global__ void update_velocities(float4* planets, float2* velocities, int num_planets){

  int thread_id = blockIdx.x * blockDim.x + threadIdx.x;
  float4 my_planet = planets[thread_id];
  // Shared memory for this block
  __shared__ float4 shared_planets[BLOCK_SIZE];

  // Compute the velocity change for planets for one block at a time
  for(int i = 0; i < num_planets; i+=blockDim.x){
    shared_planets[threadIdx.x] = planets[i + threadIdx.x];
    __syncthreads();
    float2 tempv = calculate_velocity_change_block(my_planet, shared_planets);
    velocities[thread_id].x += tempv.x;
    velocities[thread_id].y += tempv.y;
    __syncthreads();
  }
}

// TODO 7. Update the positions of the planets using the new velocities
__global__ void update_positions(float4* planets, float2* velocities, int num_planets){
  int tid = threadIdx.x + blockIdx.x * blockDim.x;
  planets[tid].x += velocities[tid].x * dT;
  planets[tid].y += velocities[tid].y * dT;
}


int main(int argc, char** argv){

    parse_args(argc, argv);
    read_planets();

    double calculation_time = 0;
    double memcopy_time = 0;



    // TODO 1. Allocate device memory, and transfer data to device

    hipMalloc(&planets_d,    sizeof(float4)*num_planets);
    hipMalloc(&velocities_d, sizeof(float2)*num_planets);

    double mem_start = walltime();
    hipMemcpy(planets_d,    planets,     sizeof(float4)*num_planets,
                                          hipMemcpyHostToDevice);
    hipMemcpy(velocities_d, velocities,  sizeof(float2)*num_planets,
                                          hipMemcpyHostToDevice);
    memcopy_time += walltime() - mem_start;


    // Calculating the number of blocks
    int num_blocks = num_planets/BLOCK_SIZE + ((num_planets%BLOCK_SIZE == 0) ? 0 : 1);


    // Main loop

    double calc_start = walltime();
    for(int t = 0; t < num_timesteps; t++){
        // TODO 2. Call kernels
        //Update velocities
        update_velocities<<<num_blocks, BLOCK_SIZE>>>(planets_d, velocities_d,
                                                      num_planets);
        //Update positions
        update_positions<<<num_blocks, BLOCK_SIZE>>>(planets_d, velocities_d,
                                                    num_planets);
    }
    calculation_time = walltime() - calc_start;



    double mem_start = walltime();
    // TODO 3. Transfer data back to host
    hipMemcpy(planets,     planets_d,    sizeof(float4)*num_planets,
                                          hipMemcpyDeviceToHost);
    hipMemcpy(velocities,  velocities_d, sizeof(float2)*num_planets,
                                          hipMemcpyDeviceToHost);
    memcopy_time += walltime() - mem_start;


    hipFree(planets_d);
    hipFree(velocities_d);
    // Output
    write_planets(num_timesteps);
    printf("%7.7f ms\n", calculation_time);
    printf("%7.7f ms\n", memcopy_time);



}
