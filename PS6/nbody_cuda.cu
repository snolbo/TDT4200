#include <stdio.h>
#include <stdlib.h>
#include <math.h>
#include <hip/hip_runtime.h>


#define dT 0.2f
#define G 0.6f
#define BLOCK_SIZE 64

// Global variables
int num_planets;
int num_timesteps;

// Host arrays
float2* velocities;
float4* planets;

// Device arrays
float2* velocities_d;
float4* planets_d;


// Parse command line arguments
void parse_args(int argc, char** argv){
    if(argc != 2){
        printf("Useage: nbody num_timesteps\n");
        exit(-1);
    }

    num_timesteps = strtol(argv[1], 0, 10);
}

// Reads planets from planets.txt
void read_planets(){

    FILE* file = fopen("planets.txt", "r");
    if(file == NULL){
        printf("'planets.txt' not found. Exiting\n");
        exit(-1);
    }

    char line[200];
    fgets(line, 200, file);
    sscanf(line, "%d", &num_planets);

    planets = (float4*)malloc(sizeof(float4)*num_planets);
    velocities = (float2*)malloc(sizeof(float2)*num_planets);

    for(int p = 0; p < num_planets; p++){
        fgets(line, 200, file);
        sscanf(line, "%f %f %f %f %f",
                &planets[p].x,
                &planets[p].y,
                &velocities[p].x,
                &velocities[p].y,
                &planets[p].z);
    }

    fclose(file);
}

// Writes planets to file
void write_planets(int timestep){
    char name[20];
    int n = sprintf(name, "planets_out.txt");

    FILE* file = fopen(name, "wr+");

    for(int p = 0; p < num_planets; p++){
        fprintf(file, "%f %f %f %f %f\n",
                planets[p].x,
                planets[p].y,
                velocities[p].x,
                velocities[p].y,
                planets[p].z);
    }

    fclose(file);
}

// TODO 7. Calculate the change in velocity for p, caused by the interaction with q
__device__ float2 calculate_velocity_change_planet(float4 p, float4 q){
  float2 dv;

  float2 dist;
  dist.x = q.x - p.x;
  dist.y = q.y - p.y;

  float abs_dist= sqrt(dist.x*dist.x + dist.y*dist.y);
  float dist_cubed = abs_dist*abs_dist*abs_dist;

  dv.x = dT*G*q.z/dist_cubed * dist.x;
  dv.y = dT*G*q.z/dist_cubed * dist.y;

  return dv;
}

// TODO 5. Calculate the change in velocity for my_planet, caused by the interactions with a block of planets
__device__ float2 calculate_velocity_change_block(float4 my_planet, float4* shared_planets){
  float2 velocity = {0.0f, 0.0f};
  for(int i = 0; i < blockDim.x; i++){
    float2 temp_vel = calculate_velocity_change_planet(my_planet, shared_planets[i]);
    velocity.x += temp_vel.x;
    velocity.y += temp_vel.y;
  }
  return velocity;
}

// TODO 4. Update the velocities by calculating the planet interactions
__global__ void update_velocities(float4* planets, float2* velocities, int num_planets){

  int tid = threadIdx.x + blockIdx.x * blockDim.x;
  float4 planet = planets[tid];
  __shared__ float4 shared_planets[BLOCK_SIZE];

  // Compute the velocity change for planets form one block at a time
  for(int i = 0; i < num_planets; i+= blockDim.x){
    shared_planets[threadIdx.x] = planets[i+threadIdx.x];
    __syncthreads();
    float2 temp_vel = calculate_velocity_change_block(planet, shared_planets);

    velocities[tid].x = temp_vel.x;
    velocities[tid].y = temp_vel.y;
  }
}

// TODO 7. Update the positions of the planets using the new velocities
__global__ void update_positions(float4* planets, float2* velocities, int num_planets){
  int tid = threadIdx.x + blockIdx.x * blockDim.x;
  planets[tid].x = velocities[tid].x * dT;
  planets[tid].y = velocities[tid].y * dT;
}


int main(int argc, char** argv){

    parse_args(argc, argv);
    read_planets();

    // TODO 1. Allocate device memory, and transfer data to device
    hipMalloc(&planets_d,    sizeof(float4)*num_planets);
    hipMalloc(&velocities_d, sizeof(float2)*num_planets);

    hipMemcpy(planets_d,    planets,     sizeof(float4)*num_planets,
                                          hipMemcpyHostToDevice);
    hipMemcpy(velocities_d, velocities,  sizeof(float2)*num_planets,
                                          hipMemcpyHostToDevice);


    // Calculating the number of blocks
    int num_blocks = num_planets/BLOCK_SIZE + ((num_planets%BLOCK_SIZE == 0) ? 0 : 1);

    // Main loop
    for(int t = 0; t < num_timesteps; t++){
        // TODO 2. Call kernels
        //Update velocities
        update_velocities<<<num_blocks, BLOCK_SIZE>>>(planets_d, velocities_d,
                                                      num_planets);
        //Update positions
        update_positions<<<num_blocks, BLOCK_SIZE>>>(planets_d, velocities_d,
                                                    num_planets);
    }

    // TODO 3. Transfer data back to host
    hipMemcpy(planets,     planets_d,    sizeof(float4)*num_planets,
                                          hipMemcpyDeviceToHost);
    hipMemcpy(velocities,  velocities_d, sizeof(float)*num_planets,
                                          hipMemcpyDeviceToHost);
    // Output
    write_planets(num_timesteps);
}
